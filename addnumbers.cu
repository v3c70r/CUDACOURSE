
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void add2(int *a)
{
    *a = *a + 2;
}

int main( void )
{
    int *data_h, *data_d;
    hipMalloc( (void**)&data_d, sizeof(int));
    //data_d = (int *)malloc(sizeof(int));
    data_h = (int *)malloc(sizeof(int));
    
    *data_h = 5;
    hipMemcpy( data_d, data_h, sizeof(int), hipMemcpyHostToDevice );
    add2<<<1,1>>>(data_d);
    hipMemcpy(data_h,	data_d, sizeof(int), hipMemcpyDeviceToHost );
    printf("data: %d\n", *data_h);
    hipFree(data_d); 
    free(data_h);
    return 0;    
}

//Res = 7
