
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void add2(int *a)
{
    *a = *a + 2;
}

int main( void )
{
    int *data;
    hipMallocManaged(&data, sizeof(int));
    
    *data = 5;
    add2<<<1,1>>>(data);
    hipDeviceSynchronize();
    printf("data: %d\n", *data);
    hipFree(data);
    return 0;    
}