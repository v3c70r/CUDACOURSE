
#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 33 * 1024;
const int threadsPerBlock = 256; 
const int blocksPerGrid = ( (N+threadsPerBlock-1) / threadsPerBlock );

static void HandleError( hipError_t err )
{
  if (err != hipSuccess) {
    printf( "%s \n", hipGetErrorString( err ));
    exit( 1 );
  }
}

__global__ void dot( float *a, float *b, float *c ) 
//Computes the dot product of vectors a and b.
//Output: c is a vector representing the dot product
//of components assigned to a block of CUDA threads
{ 
//WKSHP: Implement the dot product kernel
    int outIdx = blockIdx.x;

    __shared__ float res[threadsPerBlock];

    res[threadIdx.x] = a[threadIdx.x] * b[threadIdx.x];
    __syncthreads();

    c[outIdx] += res[threadIdx.x];
}


int main( void ) {
  float *a, *b, c, *partial_c;
  float *dev_a, *dev_b, *dev_partial_c;

  // allocate memory on the CPU side
  a = (float*)malloc( N*sizeof(float) );
  b = (float*)malloc( N*sizeof(float) );
  partial_c = (float*)malloc( blocksPerGrid*sizeof(float) );

  // allocate the memory on the GPU
  HandleError( hipMalloc( (void**)&dev_a, 
			    N*sizeof(float) ) );
  HandleError( hipMalloc( (void**)&dev_b,
			    N*sizeof(float) ) );
  HandleError( hipMalloc( (void**)&dev_partial_c, 
			    blocksPerGrid*sizeof(float) ) );

  // fill in the host memory with data
  for (int i=0; i<N; i++) 
  { 
    a[i] = i;
    b[i] = i*2; 
  }

  // copy the arrays ‘a’ and ‘b’ to the GPU
  HandleError( hipMemcpy( dev_a, a, N*sizeof(float), 
			    hipMemcpyHostToDevice ) );
  HandleError( hipMemcpy( dev_b, b, N*sizeof(float), 
			    hipMemcpyHostToDevice ) );
  dot<<<blocksPerGrid,threadsPerBlock>>>( dev_a, dev_b,
						   dev_partial_c );


 // copy the array 'c' back from the GPU to the CPU
 HandleError( hipMemcpy( partial_c, dev_partial_c, 
			   blocksPerGrid*sizeof(float),
			   hipMemcpyDeviceToHost ) );

  c = 0;
 for (int i=0; i<blocksPerGrid; i++) 
 {
   c += partial_c[i];
 }

 #define sum_squares(x) (x*(x+1)*(2*x+1)/6) 
 printf( "Does GPU value %.6g = %.6g ?\n", c,
	2 * sum_squares( (float)(N - 1) ) );


 // free memory on the GPU side
 HandleError( hipFree( dev_a ) );
 HandleError( hipFree( dev_b ) );
 HandleError( hipFree( dev_partial_c ) );

 // free memory on the CPU side
 free( a );
 free( b );
 free( partial_c );
}
