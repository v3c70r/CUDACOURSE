//errorcheck.cu - The program is designed to produce output 'data = 7'
//however, errors have been intentionally placed in the program 
//as an error checking exercise. 

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void setData(int *ptr)
{
  *ptr = 7;
}

void checkError(const char *info="")
{
    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        printf("CUDA error at %s: %s\n", info, hipGetErrorString(error));
        exit(0);
    }
}


int main(void)
{
    

    int *data_d = 0;
    int *data_h = 0;
    hipError_t error;


    error = hipMalloc((void**)&data_d, 1*sizeof(int));
    if (error != hipSuccess)
    {
        printf("pos1 CUDA error: %s\n", hipGetErrorString(error));
    }
    data_h = (int *)malloc(sizeof(int));

    setData<<<1,1>>>(data_d);

    checkError("malloc");
    hipMemcpy(data_h, data_d, sizeof(int), hipMemcpyDeviceToHost);

    checkError("memcpy");
    printf("data = %d\n", *data_h);
    free(data_h);
    hipFree(data_d);

    checkError("free memory");

    return 0;

}
