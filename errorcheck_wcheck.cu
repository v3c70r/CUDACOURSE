#include "hip/hip_runtime.h"
//errorcheck_wcheck.cu: The program is designed to produce output
//'data = 7'. However, errors have been intentionally placed into 
//the program as an exercise in error checking.
#include <stdio.h>
#include <stdlib.h>

__global__ void setData(int *ptr)
{
  *ptr = 7;
}


int main(void)
{
  int *data_d = 0;
  int *data_h = 0;
  hipError_t error;
  error = hipMalloc((void**)&data_d, UINT_MAX*sizeof(int));
  if( error != hipSuccess)
  {
    printf("hipMalloc error: %s\n", hipGetErrorString(error));
  }
  
  data_h = (int *)malloc(sizeof(int));

  setData<<<1,1>>>(0);
  hipDeviceSynchronize();
  error = hipGetLastError();
  if(error != hipSuccess)
  {
    printf("setData error: %s\n", hipGetErrorString(error));
  }
  error = hipMemcpy(data_h, data_d, sizeof(int), hipMemcpyDeviceToHost);
  if(error != hipSuccess)
  {
    printf("hipMemcpy error: %s\n", hipGetErrorString(error));
  }
  printf("data = %d\n", *data_h);
  free(data_h);
  hipFree(data_d);
  return 0;
}
