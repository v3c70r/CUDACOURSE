// includes, system

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <stdlib.h> 
 
//WORKSHOP: Change this function to a CUDA kernel
void fillArray(int *data, int N)
{
  int i;
  for( i = 0; i < N; i++)
    {
      data[i] = i;
    }
}

__global__ void fillArray(int *data, int *res)
{
    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    res[idx] = data[idx]+idx;
}
__global__ void fillArrayUnified(int *data )
{
    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    data[idx] = data[idx]+idx;
}
/////////////////////////////////////////////////////////////////////
// Program main
/////////////////////////////////////////////////////////////////////
int main( int argc, char** argv)
{
  //WORKSHOP: Declare data pointers for host and device arrays
  // (not necessary if using Unified memory)
    int *data;
    int i;
    const int N = 100;

 
 /*-----------------------------------------*/
    // allocate host memory
    //data = ( int* ) malloc(N * sizeof(int));
    ////WORKSHOP: Allocate device memory
    //// Remove the host allocation above and use cudaMallocManaged() 
    //// to allocate on host and device if using unified memory
 
    ////Fill the array
    ////WORKSHOP: Change this function call to a CUDA kernel call
    //int *d_data;
    //cudaMalloc(&d_data, sizeof(int)*N);
    //int *d_res;
    //cudaMalloc(&d_res, sizeof(int)*N);
    //cudaMemcpy(d_data, data, sizeof(int)*N, cudaMemcpyHostToDevice);
    ////int blockSize=10;
    //fillArray<<<10,10>>>(d_data, d_res);
    //cudaMemcpy(data, d_res, sizeof(float)*N, cudaMemcpyDeviceToHost);
    //cudaFree(d_res);
    //cudaFree(d_data);




/*-----------Unified memory-----------*/
    hipMallocManaged(&data, sizeof(int)*N);
    fillArrayUnified<<<10,10>>>(data );
    hipDeviceSynchronize();

    


    //WORKSHOP: Make sure the device has finished
    //WORKSHOP: Copy the results to the host
    // (not necessary if using unified memory)

    // verify the data is correct
    for (i = 0; i < N; i++)
    {
        assert(data[i] == i );
    }

    
 
    // If the program makes it this far, then the results are
    // correct and there are no run-time errors.  Good work!
    printf("Correct!\n");
 

    //Free by cuda
    hipFree(data);
    //WORKSHOP: Free the device memory
    // (if using unified memory, you can free the host and device
    //  memory with one cudaFree() call)
    return 0;
}

