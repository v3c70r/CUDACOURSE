// includes, system

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <stdlib.h> 
 

__global__ void copy(float *data_in, float *data_out, int n)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = 0;
  int xid = tid + offset;
  data_in[xid] = data_out[xid];
}
/////////////////////////////////////////////////////////////////////
// Program main
/////////////////////////////////////////////////////////////////////
int main( int argc, char** argv)
{
  float *idata_h, *odata_h;
  float *idata_d, *odata_d;
  const int N = 1000;
  hipEvent_t start, stop;
  float time, effBandwidth;
  int i;
 
    // allocate host and device memory
    idata_h = ( float* ) malloc(N * sizeof(float));
    odata_h = ( float* ) malloc(N * sizeof(float));
    hipMalloc(&idata_d, N * sizeof(float));
    hipMalloc(&odata_d, N * sizeof(float));
 
    //Fill the input array
    for (i = 0; i < N; i++)
      {
	idata_h[i] = (float) i;
      }

    //Copy the input array to the device
    hipMemcpy(idata_d, idata_h, N*sizeof(float), hipMemcpyHostToDevice);

    //Set up the timing variables and begin timing
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    //Copy Kernal
    copy<<<N, 1>>>(idata_d, idata_h, N);

    //Stop timing
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);


    //Copy the output array from the device
    hipMemcpy(odata_h, odata_d, N*sizeof(float), hipMemcpyDeviceToHost);

    // verify the data is correct
    for (i = 0; i < N; i++)
    {
        assert(odata_h[i] == idata_h[i] );
    }
 
    // If the program makes it this far, then the results are
    // correct and there are no run-time errors.  Good work!
    printf("Correct!\n");
 
    //Compute the Effective Bandwidth
    hipEventElapsedTime(&time, start, stop);
    effBandwidth = 2*N*sizeof(float)/1.0e9/time;

    printf("Kernel time = %es\n", time);
    printf("Effective Bandwidth = %e s\n", effBandwidth);
     
    //Free the device and host memory
    free(idata_h); free(odata_h);
    hipFree(idata_d); hipFree(odata_d);
    return 0;
}

